#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../src/one_qubit_simulator.h" // Include the header file

// Test function to check the initialization of the qubit
void testInitializeQubit() {
    Complex* d_state;
    hipMalloc((void**)&d_state, 2 * sizeof(Complex));

    initializeQubit<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    Complex h_state[2];
    hipMemcpy(h_state, d_state, 2 * sizeof(Complex), hipMemcpyDeviceToHost);

    assert(hipCreal(h_state[0]) == 1.0 && hipCimag(h_state[0]) == 0.0);
    assert(hipCreal(h_state[1]) == 0.0 && hipCimag(h_state[1]) == 0.0);

    hipFree(d_state);
    std::cout << "testInitializeQubit passed!" << std::endl;
}

// Test function to check the application of the Hadamard gate
void testApplyHadamard() {
    Complex* d_state;
    hipMalloc((void**)&d_state, 2 * sizeof(Complex));

    initializeQubit<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    applyHadamard<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    Complex h_state[2];
    hipMemcpy(h_state, d_state, 2 * sizeof(Complex), hipMemcpyDeviceToHost);

    double sqrt2_inv = 1.0 / sqrt(2.0);
    assert(fabs(hipCreal(h_state[0]) - sqrt2_inv) < 1e-6);
    assert(fabs(hipCimag(h_state[0]) - 0.0) < 1e-6);
    assert(fabs(hipCreal(h_state[1]) - sqrt2_inv) < 1e-6);
    assert(fabs(hipCimag(h_state[1]) - 0.0) < 1e-6);

    hipFree(d_state);
    std::cout << "testApplyHadamard passed!" << std::endl;
}

// Test that double application of X gate is identity
void testApplyX() {
    Complex* d_state;
    hipMalloc((void**)&d_state, 2 * sizeof(Complex));

    initializeQubit<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    applyX<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    applyX<<<1, 1>>>(d_state);
    hipDeviceSynchronize();

    Complex h_state[2];
    hipMemcpy(h_state, d_state, 2 * sizeof(Complex), hipMemcpyDeviceToHost);

    assert(hipCreal(h_state[0]) == 1.0 && hipCimag(h_state[0]) == 0.0);
    assert(hipCreal(h_state[1]) == 0.0 && hipCimag(h_state[1]) == 0.0);

    hipFree(d_state);
    std::cout << "testApplyX passed!" << std::endl;
}


int main() {
    testInitializeQubit();
    testApplyHadamard();
    testApplyX();
    return 0;
}